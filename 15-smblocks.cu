
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addArraysInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  printf("%d) ThIdx.x=%d, BlkIdx.x=%d, BlkDim=%d, stride=%d \n",index, threadIdx.x, blockIdx.x, blockDim.x, stride);

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;

  /*
   * Grid sizes that are multiples of the number of available SMs can
   * increase performance.
   */

  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addArraysErr;
  hipError_t asyncErr;

  printf("numberOfBlocks=%d,  threadsPerBlock=%d, N=%d\n",numberOfBlocks, threadsPerBlock, N);
  addArraysInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addArraysErr = hipGetLastError();
  if(addArraysErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addArraysErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}